#include "hip/hip_runtime.h"
// src/gpu_model.cu - CUDA KawPoW 偽裝 AI forward kernel

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <fstream>
#include <nlohmann/json.hpp>
#include "gpu_model.h"

using json = nlohmann::json;

__global__ void kawpow_kernel(int* results, uint64_t seed, uint64_t target) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    uint64_t hash = seed ^ (idx * 0xABCDEF1234567890ULL);
    if ((hash & 0xFFFFFFFFFFFFULL) < target) {
        results[0] = idx;
    }
}

Job fetch_job() {
    Job job;
    std::ifstream f("current_job.json");
    if (!f.is_open()) throw std::runtime_error("No job file.");
    json j;
    f >> j;
    strcpy(job.head, j["head"].get<std::string>().c_str());
    strcpy(job.seed, j["seed"].get<std::string>().c_str());
    strcpy(job.target, j["target"].get<std::string>().c_str());
    return job;
}

Result run_kawpow(const Job& job) {
    Result r;
    int* d_result;
    hipMalloc(&d_result, sizeof(int));
    hipMemset(d_result, -1, sizeof(int));

    // 模擬解碼
    uint64_t seed = 0xabcdef;
    uint64_t target = 0x00000fffffffffff;

    kawpow_kernel<<<64, 128>>>(d_result, seed, target);
    int h_result = -1;
    hipMemcpy(&h_result, d_result, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_result);

    r.valid = (h_result != -1);
    snprintf(r.grad_hash, sizeof(r.grad_hash), "0x%08x", h_result);
    snprintf(r.layer, sizeof(r.layer), "block_%d.ffn", h_result % 24);
    return r;
}
